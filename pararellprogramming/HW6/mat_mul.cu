#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include "mat_mul.h"
#define SMCNT  46
#define WRAPSZ 32

#define CUDA_CALL(f)                                                       \
  {                                                                        \
    hipError_t err = (f);                                                 \
    if (err != hipSuccess) {                                              \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__, \
              err, hipGetErrorString(err));                               \
      exit(1);                                                             \
    }                                                                      \
  }

// Super slow sgemm kernel
__global__ void sgemm(float *A, float *B, float *C, int M, int N, int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N) return;

  C[i * N + j] = 0;
  for(int k = 0; k < K; k += 2){
    C[i * N + j] += A[i * K + k] * B[k * N + j];
    C[i * N + j] += A[i * K + (k + 1)] * B[(k + 1) * N + j];    
  }
}

// Device (GPU) pointers
static float *a_d;
static float *b_d;
static float *c_d;

void mat_mul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // Launch kernel on every GPU
  printf("Start mat mul ... \n");
  // int count;
  // hipGetDeviceCount(&count);
  // printf("Number of devices: %d\n", count);

  // printf("Getting dev info ... \n");
  // hipDeviceProp_t props[10];
  // for (int i = 0; i < count; ++i) {
  //   printf("\tdevice %d:\n", i);
  //   hipGetDeviceProperties(&props[i], i);
  //   printf("\t\tname: %s\n", props[i].name);
  //   printf("\t\tmultiProcessorCount: %d\n", props[i].multiProcessorCount);
  //   printf("\t\tmaxThreadsPerBlock: %d\n", props[i].maxThreadsPerBlock);
  //   printf("\t\ttotalGlobalMem: %lu\n", props[i].totalGlobalMem);
  //   printf("\t\tsharedMemPerBlock: %lu\n", props[i].sharedMemPerBlock);
  // }

  int targetBlkSz = 4;
  int blkSz = 2; int blkCnt = M * N / (blkSz * blkSz);

  if(M % targetBlkSz == 0 && N % targetBlkSz == 0 && K % targetBlkSz == 0){
    printf("optimized multiplication start ... \n"); fflush(stdout);
    blkSz = targetBlkSz;
    blkCnt = M * N / (blkSz * blkSz);  // = 8192 * 8192 / (64 * 64) = (128 * 128)

    if(blkCnt < SMCNT){
      printf("block count error !! \n"); fflush(stdout);
    }
    if((blkSz * blkSz) % WRAPSZ != 0){
      printf("block size error !! \n"); fflush(stdout);
    }
  }

  printf("block size is %d * %d = %d \n", blkSz, blkSz, blkSz * blkSz); fflush(stdout);
  printf("the number of block is %d \n", blkCnt); fflush(stdout);

  dim3 blockDim(blkSz, blkSz, 1);
  dim3 gridDim(M, N, 1);

  printf("Start sgemm \n"); fflush(stdout);
  printf("M, N, K %d %d %d \n", M, N, K);
  sgemm<<<gridDim, blockDim>>>(a_d, b_d, c_d, M, N, K);
  printf("Done sgemm \n"); fflush(stdout);

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  printf("Sync Start ... \n"); fflush(stdout);
  CUDA_CALL(hipDeviceSynchronize());
  printf("Sync Done \n"); fflush(stdout);
}

void mat_mul_init(float *A, float *B, float *C, int M, int N, int K) {
  // Allocate device memory
  // M 8196, N 8196, K 8196
  printf("mat mul init ... \n");    
  CUDA_CALL(hipMalloc(&a_d, M * K * sizeof(float)));
  CUDA_CALL(hipMalloc(&b_d, K * N * sizeof(float)));
  CUDA_CALL(hipMalloc(&c_d, M * N * sizeof(float)));

  // Upload A and B matrix to GPU
  CUDA_CALL(hipMemcpy(a_d, A, M * K * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(b_d, B, K * N * sizeof(float), hipMemcpyHostToDevice));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CUDA_CALL(hipDeviceSynchronize());
  printf("mat mul init done ! \n");   fflush(stdout);  
}

void mat_mul_final(float *A, float *B, float *C, int M, int N, int K) {
  // Do any post-matmul cleanup work here.

  printf("mat mul final ... \n");  fflush(stdout);
  // Download C matrix from GPU
  CUDA_CALL(hipMemcpy(C, c_d, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CUDA_CALL(hipDeviceSynchronize());
  printf("mat mul final done ! \n");     fflush(stdout);
}
