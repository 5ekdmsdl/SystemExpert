
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void vec_add_kernel(const int *A, const int *B, int *C, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) C[i] = A[i] + B[i];
}

__global__ void kernel_add(const int* a, const int* b, int* c){
  *c = *a + *b;
}

int main() {
  // int N = 16384;    // = 32 * 512
  // int *A = (int *) malloc(N * sizeof(int));
  // int *B = (int *) malloc(N * sizeof(int));
  // int *C = (int *) malloc(N * sizeof(int));
  // int *C_ans = (int *) malloc(N * sizeof(int));

  // for (int i = 0; i < N; i++) {
  //   A[i] = rand() % 1000;
  //   B[i] = rand() % 1000;
  //   C_ans[i] = A[i] + B[i];
  // }

  // TODO: Run vector addition on GPU
  // Save the result in C
  int a = 1, b = 2, c;
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, sizeof(int));
  hipMalloc(&d_b, sizeof(int));
  hipMalloc(&d_c, sizeof(int));

  hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
  kernel_add<<<1,1>>>(d_a, d_b, d_c);
  hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  printf("c : %d \n",c);


  // for (int i = 0; i < N; i++) {
  //   if (C[i] != C_ans[i]) {
  //     printf("Result differ at %d: %d vs %d\n", i, C[i], C_ans[i]);
  //   }
  // }

  printf("Validation done.\n");

  return 0;
}
