#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  // TODO
  int count;
  cudeGetDeviceCount(&count);

  hipDeviceProp_t prop[4];
  for (int i = 0; i < count; i++) {
    hipGetDeviceProperties(&prop[i], i);
    printf("\t\tname: %s\n", props[i].name);
    printf("\t\tmultiProcessorCount: %d\n", props[i].multiProcessorCount);
    printf("\t\tmaxThreadsPerBlock: %d\n", props[i].maxThreadsPerBlock);
    printf("\t\ttotalGlobalMem: %lu\n", props[i].totalGlobalMem);
    printf("\t\tsharedMemPerBlock: %lu\n", props[i].sharedMemPerBlock);
  
  }
  



  return 0;
}
