#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#define MAX_NUM_GPU 4
int num_devices = 0;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  // FILL IN HERE
}


// Array of device (GPU) pointers
static float *a_d[MAX_NUM_GPU];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];

int devCnt = 0;

void matmul(float *A, float *B, float *C, int M, int N, int K) {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
  
  if(mpi_rank == 0){
    for (int i = 0; i < devCnt; i++) {
      hipSetDevice(i);
      hipMemcpy(a_d[i], A, sizeof(float) * M * K, hipMemcpyHostToDevice);
      hipMemcpy(b_d[i], B, sizeof(float) * K * N, hipMemcpyHostToDevice);
      printf("dev %d copy done ! \n ", i); 
    }
  }
}

void matmul_initialize(int M, int N, int K) {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  if(mpi_rank == 0){
    hipGetDeviceCount(&devCnt);

    for (int i = 0; i < devCnt; i++) {
      hipSetDevice(i);
      // hipStreamCreate(&stream[i]);
      // hipEventCreate(&events[i]);

      hipMalloc(&a_d[i], sizeof(float) * M * K);
      hipMalloc(&b_d[i], sizeof(float) * K * N);
      hipMalloc(&c_d[i], sizeof(float) * M * N);

      printf("dev %d malloc Done ! \n", i);
    }
  }
}

void matmul_finalize() {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);
  
  // FILL IN HERE
}
