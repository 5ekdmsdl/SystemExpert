#include "hip/hip_runtime.h"
#include "matmul.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <mpi.h>
#define SMCNT  46
#define WRAPSZ 32
#define ROOTRANK 0

#define CUDA_CALL(f)                                                           \
  {                                                                            \
    hipError_t err = (f);                                                     \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at [%s:%d] %d %s\n", __FILE__, __LINE__,     \
              err, hipGetErrorString(err));                                   \
      exit(1);                                                                 \
    }                                                                          \
  }

#define MAX_NUM_GPU 1
int num_devices = 0;

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  if (i >= M || j >= N) return;

  float sum = 0;
  for(int k = 0; k < K; k++){
    sum += A[i * K + k] * B[k * N + j];  
  }
  C[i * N + j] = sum;
}

// Array of device (GPU) pointers
static float *a_d[MAX_NUM_GPU];
static float *b_d[MAX_NUM_GPU];
static float *c_d[MAX_NUM_GPU];
static int Mbegin[MAX_NUM_GPU], Mend[MAX_NUM_GPU];

int devCnt = 0;

void matmul(float *A, float *B, float *C, int M, int N, int K) {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  int blkSz = 2; int blkCnt = M * N / (blkSz * blkSz);
  if(mpi_rank == ROOTRANK){
    for (int i = 0; i < num_devices; i++) {
      printf("Memcopying ... dev %d \n", i);
      hipSetDevice(i);
      CUDA_CALL(hipMemcpy(a_d[i], A, M * K * sizeof(float), hipMemcpyHostToDevice));
      CUDA_CALL(hipMemcpy(b_d[i], B, K * N * sizeof(float), hipMemcpyHostToDevice));
    
      printf("Memcopying Done ! dev %d \n", i);
    }
    
    int targetBlkSz = 8;
    if(M % targetBlkSz == 0 && N % targetBlkSz == 0 && K % targetBlkSz == 0){
      printf("optimized multiplication start ... \n"); fflush(stdout);
      blkSz = targetBlkSz;
      blkCnt = M * N / (blkSz * blkSz);  // = 8192 * 8192 / (64 * 64) = (128 * 128)

      if(blkCnt < SMCNT){
        printf("block count error !! \n"); fflush(stdout);
      }
      if((blkSz * blkSz) % WRAPSZ != 0){
        printf("block size error !! \n"); fflush(stdout);
      }  
    }      
    
    printf("block size is %d * %d = %d \n", blkSz, blkSz, blkSz * blkSz); fflush(stdout);
    printf("grid size is %d * %d \n", M / blkSz, N / blkSz); fflush(stdout);
  }
  MPI_Barrier(MPI_COMM_WORLD);
  
  if(mpi_rank == ROOTRANK){
    for (int devNum = 0; devNum < num_devices; devNum++) {
      printf("Device num : %d ... \n", devNum);
      hipSetDevice(devNum);

      dim3 blockDim(blkSz, blkSz, 1);
      dim3 gridDim(M / blkSz, N / blkSz, 1);

      printf("Start matmul_kernel \n"); fflush(stdout);
      matmul_kernel<<<gridDim, blockDim>>>(A, B, C, M, N, K);
      printf("Done matmul_kernel \n"); fflush(stdout);
    }    
  }
  MPI_Barrier(MPI_COMM_WORLD);
}

void matmul_initialize(int M, int N, int K) {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  if(mpi_rank == 0){
    hipGetDeviceCount(&devCnt);

    for (int i = 0; i < devCnt; i++) {
      hipSetDevice(i);
      // hipStreamCreate(&stream[i]);
      // hipEventCreate(&events[i]);

      hipMalloc(&a_d[i], sizeof(float) * M * K);
      hipMalloc(&b_d[i], sizeof(float) * K * N);
      hipMalloc(&c_d[i], sizeof(float) * M * N);

      printf("dev %d malloc Done ! \n", i);
    }
  }
}

void matmul_finalize() {
  int mpi_rank, mpi_world_size;
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_world_size);

  if(mpi_rank == ROOTRANK){
    printf("mat mul final ... \n");  fflush(stdout);
    printf("mat mul final done ! \n");     fflush(stdout);
  }
}
